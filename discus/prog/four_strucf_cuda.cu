
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_complex.h"

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void init_array_kernel(hipFloatComplex*, int);

__global__ void four_strucf_cuda_kernel(hipFloatComplex*, hipFloatComplex*,
					int, int, int, int, int,
					int*, int*, int*, int*);

__global__ void iin_xat(float*, int*, int*, int*, int*,
			float, float, float,
			float, float, float,
			float, float, float,
			float, float, float,
			int, int);

extern "C"{
  void four_strucf_cuda_(hipFloatComplex *tcsf, hipFloatComplex *cex,
			 float *xat, int *nxat, int *num,
			 float *xm, float *win, float *vin, float *uin, int *cr_natoms)
  {
    int nnum = num[0]*num[1]*num[2];
    int blockDim = 64;
    int gridDim = (nnum + blockDim - 1) / blockDim;
    int gridDim2 = (cr_natoms[0]+blockDim - 1 ) / blockDim;
    
    //Allocate space for complex exponent and copy to cex to device.
    hipFloatComplex* d_cex;
    hipMalloc((void**) &d_cex, I2PI * sizeof(hipFloatComplex));
    hipMemcpy(d_cex,      cex, I2PI * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
    //Allocate space for csf table and initalise to zero.
    hipFloatComplex* d_tcsf;
    hipMalloc((void**) &d_tcsf, nnum * sizeof(hipFloatComplex));
    init_array_kernel<<<gridDim, blockDim>>>(d_tcsf, nnum);
    
    //Allocate space for iarg0, iincu, iincv, iincw
    int* d_iarg0;
    int* d_iincu;
    int* d_iincv;
    int* d_iincw;
    hipMalloc((void**) &d_iarg0, nxat[0] * sizeof(int));
    hipMalloc((void**) &d_iincu, nxat[0] * sizeof(int));
    hipMalloc((void**) &d_iincv, nxat[0] * sizeof(int));
    hipMalloc((void**) &d_iincw, nxat[0] * sizeof(int));
    
    //Allocate space for list of atoms (xat) and copy to device
    float* d_xat;
    hipMalloc((void**) &d_xat, cr_natoms[0] * 3 * sizeof(float));
    hipMemcpy(d_xat,      xat, cr_natoms[0] * 3 * sizeof(float), hipMemcpyHostToDevice);
    
    iin_xat<<<gridDim2, blockDim>>>
      (d_xat, d_iarg0, d_iincu, d_iincv, d_iincw,
       xm[0], xm[1], xm[2],
       uin[0], uin[1], uin[2],
       vin[0], vin[1], vin[2],
       win[0], win[1], win[2],
       cr_natoms[0], nxat[0]);
    
    four_strucf_cuda_kernel<<<gridDim, blockDim>>>
      (d_cex, d_tcsf,
       num[0],num[1],num[2],nnum,nxat[0],
       d_iarg0, d_iincu, d_iincv, d_iincw);
    
    hipMemcpy(tcsf, d_tcsf, nnum*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    
    hipFree(d_tcsf);
    hipFree(d_cex);
    hipFree(d_xat);
    hipFree(d_iarg0);
    hipFree(d_iincu);
    hipFree(d_iincv);
    hipFree(d_iincw);
    
  }
}

__global__ void iin_xat(float* d_xat, int* d_iarg0, int* d_iincu, int* d_iincv, int* d_iincw,
			float xm1,  float xm2,  float xm3,
			float uin1, float uin2, float uin3,
			float vin1, float vin2, float vin3,
			float win1, float win2, float win3,
			int cr_natoms, int nxat)
{
  unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
  double xarg0, xincu, xincv, xincw;
  
  if(idx<nxat)
    {
      xarg0 = xm1  * d_xat[idx] + xm2  * d_xat[idx+cr_natoms] + xm3  * d_xat[idx+cr_natoms*2];
      xincu = uin1 * d_xat[idx] + uin2 * d_xat[idx+cr_natoms] + uin3 * d_xat[idx+cr_natoms*2];
      xincv = vin1 * d_xat[idx] + vin2 * d_xat[idx+cr_natoms] + vin3 * d_xat[idx+cr_natoms*2];
      xincw = win1 * d_xat[idx] + win2 * d_xat[idx+cr_natoms] + win3 * d_xat[idx+cr_natoms*2];
      d_iarg0[idx] = (int)rintf(64 * I2PI * (xarg0 - (int)xarg0 + 1.));
      d_iincu[idx] = (int)rintf(64 * I2PI * (xincu - (int)xincu + 1.));
      d_iincv[idx] = (int)rintf(64 * I2PI * (xincv - (int)xincv + 1.));
      d_iincw[idx] = (int)rintf(64 * I2PI * (xincw - (int)xincw + 1.));
    }
}

__global__ void four_strucf_cuda_kernel(hipFloatComplex* d_cex, hipFloatComplex* d_tcsf,
					int num1,   int   num2, int num3, int nnum, int nxat,
					int* d_iarg0, int* d_iincu, int* d_iincv, int* d_iincw)
{
  int i, j, k, n, iadd, iarg;
  unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
  //Loop over all image pixels 'idx'. 'iadd' is the address of the
  //argument to the complex exponent (in the table 'cex()'). The '>>6'
  //operation divides out the 64 and the '&MASK' is used so that the
  //argument to the complex exponent is inside our table which has
  //range 0=>2pi.
  
  if(idx<nnum)
    {
      i = idx / (num2 * num3);
      j = (idx / num3) % num2;
      k = idx % num3;
      for(n=0; n < nxat; n++)
	{
	  iarg = d_iarg0[n] + i * d_iincu[n] + j * d_iincv[n] + k * d_iincw[n];
	  iadd = iarg >> 6;
	  iadd = iadd & MASK;
	  d_tcsf[idx] = hipCaddf(d_tcsf[idx],d_cex[iadd]);
	}
    }
  __syncthreads();
}

__global__ void init_array_kernel(hipFloatComplex* array, int n)
{
  unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx<n)
    array[idx] = make_hipFloatComplex(0.0,0.0);
  __syncthreads();
}
