
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void initarraygpu(float[], float[], int);

__global__ void computestrucf(float, float, float,
			      float*, float*,
			      float*, float*,
			      int, int,
			      float, float, float,
			      float, float, float,
			      float, float, float,
			      float, float, float);

extern "C"{
  void cudastrucf_(float *csf_r, float *csf_i, float *cex_r, float *cex_i, float *xat, int *nxat, int *num, float *xm, float *win, float *vin, float *uin, int *cr_natoms)
  {
    int nnum = num[0]*num[1]*num[2];
    
    int threadsPerBlock = 64;
    int threadsPerGrid = (nnum + threadsPerBlock - 1) / threadsPerBlock;
    
    float* d_rtcsf;
    hipMalloc((void**) &d_rtcsf, nnum * sizeof(float));
    float* d_itcsf;
    hipMalloc((void**) &d_itcsf, nnum * sizeof(float));
    
    float* d_rexp;
    hipMalloc((void**) &d_rexp, I2PI * sizeof(float));
    float* d_iexp;
    hipMalloc((void**) &d_iexp, I2PI * sizeof(float));
    
    hipMemcpy(d_rexp, cex_r, I2PI * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_iexp, cex_i, I2PI * sizeof(float), hipMemcpyHostToDevice);
    
    initarraygpu<<<threadsPerGrid, threadsPerBlock>>>(d_rtcsf, d_itcsf, nnum);
    
    printf("Starting CUDA!\n");
    
    for(int l=0; l< nxat[0]; l++)
      {
	computestrucf<<<threadsPerGrid, threadsPerBlock>>>
	  (xat[l], xat[l+cr_natoms[0]+1], xat[l+cr_natoms[0]+2],
	   d_rexp, d_iexp,
	   d_rtcsf, d_itcsf,
	   num[0],num[1],
	   xm[0], xm[1], xm[2],
	   uin[0], uin[1], uin[2],
	   vin[0], vin[1], vin[2],
	   win[0], win[1], win[2]);
      }
    
    hipMemcpy(csf_r, d_rtcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(csf_i, d_itcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    
    
    hipFree(d_rtcsf);
    hipFree(d_itcsf);
    hipFree(d_rexp);
    hipFree(d_iexp);
    
  }
}

__global__ void computestrucf(float xat1, float xat2, float xat3, 
			      float* exp_r, float* exp_i,
			      float* tcsf_r, float* tcsf_i,
			      int num1, int num2,
			      float xm1, float xm2, float xm3,
			      float uin1, float uin2, float uin3,
			      float vin1, float vin2, float vin3, 
			      float win1, float win2, float win3)
{
  float xarg0, xincu, xincv;//, xincw;
  int iarg, iarg0, iincu, iincv;//, iincw;
  int i, j, iadd, id;
  
  xarg0 = xm1 * xat1 + xm2 * xat2 + xm3 * xat3;
  xincu = uin1 * xat1 + uin2 * xat2 + uin3 * xat3;
  xincv = vin1 * xat1 + vin2 * xat2 + vin3 * xat3;
  //xincw = win1 * xat1 + win2 * xat2 + win3 * xat3;
  iarg0 = (int)rintf(64 * I2PI * (xarg0 - (int)xarg0 + 1.));
  iincu = (int)rintf(64 * I2PI * (xincu - (int)xincu + 1.));
  iincv = (int)rintf(64 * I2PI * (xincv - (int)xincv + 1.));
  //iincw = (int)rintf(64 * I2PI * (xincw - (int)xincw + 1.));
  
  id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<num1*num2)
    {
      i = id / num1;
      j = id % num2;
      iarg = iarg0 + j * iincu + i * iincv;
      iadd = iarg >> 6;
      iadd = iadd & MASK;
      //tcsf_r[l*num1+k] += exp_r[iadd];
      //tcsf_i[l*num1+k] += exp_i[iadd];
      tcsf_r[id] += exp_r[iadd];
      tcsf_i[id] += exp_i[iadd];
    };
  __syncthreads();
}


__global__ void initarraygpu(float* array1, float* array2, int nelements)
{
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<nelements)
    {
      array1[id] = 0.0;
      array2[id] = 0.0;
    };
  __syncthreads();
}

