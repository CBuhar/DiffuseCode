
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void initarraygpu(float[], float[], int);

__global__ void computestrucf(float*, float*,
			      float*, float*,
			      int, int,
			      int, int, int);

extern "C"{
  void cudastrucf_(float *csf_r, float *csf_i, float *cex_r, float *cex_i, float *xat, int *nxat, int *num, float *xm, float *win, float *vin, float *uin, int *cr_natoms)
  {
    int nnum = num[0]*num[1]*num[2];
    
    int threadsPerBlock = 64;
    int threadsPerGrid = (nnum + threadsPerBlock - 1) / threadsPerBlock;
    
    float* d_rtcsf;
    hipMalloc((void**) &d_rtcsf, nnum * sizeof(float));
    float* d_itcsf;
    hipMalloc((void**) &d_itcsf, nnum * sizeof(float));
    
    float* d_rexp;
    hipMalloc((void**) &d_rexp, I2PI * sizeof(float));
    float* d_iexp;
    hipMalloc((void**) &d_iexp, I2PI * sizeof(float));
    
    hipMemcpy(d_rexp, cex_r, I2PI * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_iexp, cex_i, I2PI * sizeof(float), hipMemcpyHostToDevice);
    
    initarraygpu<<<threadsPerGrid, threadsPerBlock>>>(d_rtcsf, d_itcsf, nnum);
    
    printf("Starting CUDA!!\n");
    
    float xarg0, xincu, xincv;//, xincw;
    int iarg0, iincu, iincv;//, iincw;
    
    for(int l=0; l< nxat[0]; l++){
      xarg0 = xm[0] * xat[l] + xm[1] * xat[l+cr_natoms[0]+1] + xm[2] * xat[l+cr_natoms[0]+2];
      xincu = uin[0] * xat[l] + uin[1] * xat[l+cr_natoms[0]+1] + uin[2] * xat[l+cr_natoms[0]+2];
      xincv = vin[0] * xat[l] + vin[1] * xat[l+cr_natoms[0]+1] + vin[2] * xat[l+cr_natoms[0]+2];
      //xincw = win1 * xat1 + win2 * xat2 + win3 * xat3;
      iarg0 = (int)rintf(64 * I2PI * (xarg0 - (int)xarg0 + 1.));
      iincu = (int)rintf(64 * I2PI * (xincu - (int)xincu + 1.));
      iincv = (int)rintf(64 * I2PI * (xincv - (int)xincv + 1.));
      //iincw = (int)rintf(64 * I2PI * (xincw - (int)xincw + 1.));
      
      computestrucf<<<threadsPerGrid, threadsPerBlock>>>
	(d_rexp, d_iexp,
	 d_rtcsf, d_itcsf,
	 num[0],num[1],
	 iarg0,iincu,iincv);
    }
    
    hipMemcpy(csf_r, d_rtcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(csf_i, d_itcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    
    
    hipFree(d_rtcsf);
    hipFree(d_itcsf);
    hipFree(d_rexp);
    hipFree(d_iexp);
    
  }
}

__global__ void computestrucf(float* exp_r, float* exp_i,
			      float* tcsf_r, float* tcsf_i,
			      int num1, int num2,
			      int iarg0, int iincu, int iincv)
{
  int i, j, iadd, id, iarg;
  
  id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<num1*num2)
    {
      i = id / num1;
      j = id % num1;
      iarg = iarg0 + j * iincu + i * iincv;
      iadd = iarg >> 6;
      iadd = iadd & MASK;
      //tcsf_r[i*num1+j] += exp_r[iadd];
      //tcsf_i[i*num1+j] += exp_i[iadd];
      tcsf_r[id] += exp_r[iadd];
      tcsf_i[id] += exp_i[iadd];
    };
  __syncthreads();
}


__global__ void initarraygpu(float* array1, float* array2, int nelements)
{
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<nelements)
    {
      array1[id] = 0.0;
      array2[id] = 0.0;
    };
  __syncthreads();
}

