
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_complex.h"

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void init_array_kernel(hipFloatComplex*, int);

__global__ void four_strucf_cuda_kernel(hipFloatComplex*, hipFloatComplex*,
					int, int, int,
					int, int, int);

extern "C"{
  void four_strucf_cuda_(hipFloatComplex *tcsf, hipFloatComplex *cex, float *xat, int *nxat, int *num, float *xm, float *win, float *vin, float *uin, int *cr_natoms)
  {
    int nnum = num[0]*num[1]*num[2];
    double xarg0, xincu, xincv;
    int iarg0, iincu, iincv;
    int blockDim = 64;
    int gridDim = (nnum + blockDim - 1) / blockDim;
    
    //Allocate space for complex exponent and copy to cex to device.
    hipFloatComplex* d_cex;
    hipMalloc((void**) &d_cex, I2PI * sizeof(hipFloatComplex));
    hipMemcpy(d_cex, cex, I2PI * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
    //Allocate space for csf table and initalise to zero.
    hipFloatComplex* d_tcsf;
    hipMalloc((void**) &d_tcsf, nnum * sizeof(hipFloatComplex));
    init_array_kernel<<<gridDim, blockDim>>>(d_tcsf, nnum);
    
    //Loop over all of the atoms we are handling now ... 
    for(int l=0; l< nxat[0]; l++){
      // Get initial argument to the exponent and increments along the two axies 'u' and 'v'.
      xarg0 = xm[0]  * xat[l] + xm[1]  * xat[l+cr_natoms[0]] + xm[2]  * xat[l+cr_natoms[0]*2];
      xincu = uin[0] * xat[l] + uin[1] * xat[l+cr_natoms[0]] + uin[2] * xat[l+cr_natoms[0]*2];
      xincv = vin[0] * xat[l] + vin[1] * xat[l+cr_natoms[0]] + vin[2] * xat[l+cr_natoms[0]*2];
      //Convert to high precision integers (64*i2pi=2^20) ...
      iarg0 = (int) rint( 64 * I2PI * (xarg0 - (int) xarg0 + 1.) );
      iincu = (int) rint( 64 * I2PI * (xincu - (int) xincu + 1.) );
      iincv = (int) rint( 64 * I2PI * (xincv - (int) xincv + 1.) );
      
      four_strucf_cuda_kernel<<<gridDim, blockDim>>>
	(d_cex, d_tcsf,
	 num[0],num[1],nnum,
	 iarg0,iincu,iincv);
    }
    
    hipMemcpy(tcsf, d_tcsf, nnum*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    
    hipFree(d_tcsf);
    hipFree(d_cex);
    
  }
}

__global__ void four_strucf_cuda_kernel(hipFloatComplex* d_cex, hipFloatComplex* d_tcsf,
					int num1, int num2, int nnum,
					int iarg0, int iincu, int iincv)
{
  int i, j, iadd, iarg;
  unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
  //Loop over all image pixels 'idx'. 'iadd' is the address of the
  //argument to the complex exponent (in the table 'cex()'). The '>>6'
  //operation divides out the 64 and the '&MASK' is used so that the
  //argument to the complex exponent is inside our table which has
  //range 0=>2pi.
  if(idx<nnum)
    {
      i = idx / num1;
      j = idx % num1;
      iarg = iarg0 + i * iincu + j * iincv;
      iadd = iarg >> 6;
      iadd = iadd & MASK;
      d_tcsf[idx] = hipCaddf(d_tcsf[idx],d_cex[iadd]);
    };
  __syncthreads();
}

__global__ void init_array_kernel(hipFloatComplex* array, int n)
{
  unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx<n)
    array[idx] = make_hipFloatComplex(0.0,0.0);
  __syncthreads();
}
