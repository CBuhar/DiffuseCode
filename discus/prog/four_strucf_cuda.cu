
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define I2PI (1<<16)
#define MASK (I2PI-1)

__global__ void initarraygpu(float[], float[], int);

__global__ void computestrucf(float, float, float, float[], float[], float[], float[],
			      int, int, float, float, float,
			      float, float, float, float, float, float, 
			      float, float, float);

extern "C" int __config_mod_MOD_nmax;
extern "C" float __diffuse_mod_MOD_xm[3], __diffuse_mod_MOD_win[3], __diffuse_mod_MOD_vin[3], __diffuse_mod_MOD_uin[3];
extern "C" int  __diffuse_mod_MOD_num[3], __diffuse_mod_MOD_nxat;
extern "C" int __crystal_mod_MOD_cr_natoms;

extern "C"{
  void cudastrucf_(float *csf_r, float *csf_i, float *cex_r, float *cex_i, float *xat)
  {
    int nnum = __diffuse_mod_MOD_num[0]*__diffuse_mod_MOD_num[1]*__diffuse_mod_MOD_num[2];
    
    int threadsPerBlock = 64;
    int threadsPerGrid = (nnum + threadsPerBlock - 1) / threadsPerBlock;
    
    float* d_rtcsf;
    hipMalloc((void**) &d_rtcsf, nnum * sizeof(float));
    float* d_itcsf;
    hipMalloc((void**) &d_itcsf, nnum * sizeof(float));
    
    float* d_rexp;
    hipMalloc((void**) &d_rexp, I2PI * sizeof(float));
    float* d_iexp;
    hipMalloc((void**) &d_iexp, I2PI * sizeof(float));
    
    hipMemcpy(d_rexp, cex_r, I2PI * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_iexp, cex_i, I2PI * sizeof(float), hipMemcpyHostToDevice);
    
    initarraygpu<<<threadsPerGrid, threadsPerBlock>>>(d_rtcsf, d_itcsf, nnum);
    
    for(int l=0; l< __diffuse_mod_MOD_nxat; l++)
      {
	computestrucf<<<threadsPerGrid, threadsPerBlock>>>(xat[l], xat[l+__crystal_mod_MOD_cr_natoms+1], xat[l+__crystal_mod_MOD_cr_natoms+2], d_rexp, d_iexp, d_rtcsf, d_itcsf, __diffuse_mod_MOD_num[0], __diffuse_mod_MOD_num[1], __diffuse_mod_MOD_xm[0], __diffuse_mod_MOD_xm[1], __diffuse_mod_MOD_xm[2], __diffuse_mod_MOD_uin[0], __diffuse_mod_MOD_uin[1], __diffuse_mod_MOD_uin[2], __diffuse_mod_MOD_vin[0], __diffuse_mod_MOD_vin[1], __diffuse_mod_MOD_vin[2], __diffuse_mod_MOD_win[0], __diffuse_mod_MOD_win[1], __diffuse_mod_MOD_win[2]);
      }
    
    hipMemcpy(csf_r, d_rtcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(csf_i, d_itcsf, nnum*sizeof(float), hipMemcpyDeviceToHost);
    
    
    hipFree(d_rtcsf);
    hipFree(d_itcsf);
    hipFree(d_rexp);
    hipFree(d_iexp);
    
  }
}

__global__ void computestrucf(float xat1, float xat2, float xat3, 
			      float* exp_r, float* exp_i,
			      float* tcsf_r, float* tcsf_i,
			      int num1, int num2,
			      float xm1, float xm2, float xm3,
			      float uin1, float uin2, float uin3,
			      float vin1, float vin2, float vin3, 
			      float win1, float win2, float win3)
{
  float xarg0, xincu, xincv;//, xincw;
  int iarg, iarg0, iincu, iincv;//, iincw;
  int i, j, iadd, id;
  
  xarg0 = xm1 * xat1 + xm2 * xat2 + xm3 * xat3;
  xincu = uin1 * xat1 + uin2 * xat2 + uin3 * xat3;
  xincv = vin1 * xat1 + vin2 * xat2 + vin3 * xat3;
  //xincw = win1 * xat1 + win2 * xat2 + win3 * xat3;
  iarg0 = (int)rintf(64 * I2PI * (xarg0 - (int)xarg0 + 1.));
  iincu = (int)rintf(64 * I2PI * (xincu - (int)xincu + 1.));
  iincv = (int)rintf(64 * I2PI * (xincv - (int)xincv + 1.));
  //iincw = (int)rintf(64 * I2PI * (xincw - (int)xincw + 1.));
  
  id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<num1*num2)
    {
      i = id / num1;
      j = id % num2;
      iarg = iarg0 + j * iincu + i * iincv;
      iadd = iarg >> 6;
      iadd = iadd & MASK;
      //tcsf_r[l*num1+k] += exp_r[iadd];
      //tcsf_i[l*num1+k] += exp_i[iadd];
      tcsf_r[id] += exp_r[iadd];
      tcsf_i[id] += exp_i[iadd];
    };
  __syncthreads();
}


__global__ void initarraygpu(float* array1, float* array2, int nelements)
{
  int id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id<nelements)
    {
      array1[id] = 0.0;
      array2[id] = 0.0;
    };
  __syncthreads();
}

